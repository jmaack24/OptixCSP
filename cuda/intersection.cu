#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/helpers.h>
#include "Soltrace.h"
#include <stdio.h>
#include <optix_device.h>
#include <sutil/vec_math.h>
#include "GeometryDataST.h"

extern "C" __global__ void __intersection__parallelogram()
{
    // Load shader binding table (SBT) and access data specific to this hit group
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryDataST::Parallelogram& parallelogram = sbt_data->geometry_data.getParallelogram();

    // Get ray information: origin, direction, and min/max distances over which ray should be tested
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float  ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    // Compute ray intersection point
    float3 n  = make_float3( parallelogram.plane );
    float  dt = dot( ray_dir, n );
    // Compute distance t (point of intersection) along ray direction from ray origin
    float  t  = ( parallelogram.plane.w - dot( n, ray_orig ) ) / dt;

    // Verify intersection distance and Report ray intersection point
    if( t > ray_tmin && t < ray_tmax )
    {
        float3 p  = ray_orig + ray_dir * t;
        float3 vi = p - parallelogram.anchor;
        float  a1 = dot( parallelogram.v1, vi );
        if( a1 >= 0 && a1 <= 1 )
        {
            float a2 = dot( parallelogram.v2, vi );
            if( a2 >= 0 && a2 <= 1 )
            {
                // float as uint? is this a boolean? 
                optixReportIntersection( t, 0, float3_as_args( n ), __float_as_uint( a1 ), __float_as_uint( a2 ) );
            }
        }
    }
}

extern "C" __global__ void __intersection__rectangle_flat()
{

    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryDataST::Rectangle_Flat& rectangle = sbt_data->geometry_data.getRectangle_Flat();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float ray_tmin = optixGetRayTmin();
    const float ray_tmax = optixGetRayTmax();

    // Get plane normal and distance
    float3 n = make_float3(rectangle.plane);
    float dt = dot(ray_dir, n);
    
    // Compute distance t (point of intersection) along ray direction from ray origin
    float t = (rectangle.plane.w - dot(n, ray_orig)) / dt;

    // Verify intersection distance
    if (t > ray_tmin && t < ray_tmax)
    {
        // Compute intersection point
        float3 p = ray_orig + ray_dir * t;
        
        // Compute vector from center to intersection point
        float3 v = p - rectangle.center;
        
        // Project onto x and y to get local coordinates
        float x = dot(rectangle.x, v);
        float y = dot(rectangle.y, v);
        
        // Check if point is within rectangle bounds
        if (x >= -rectangle.width/2 && x <= rectangle.width/2 &&
            y >= -rectangle.height/2 && y <= rectangle.height/2)
        {
            // Use raw coordinates like parallelogram intersection
            optixReportIntersection(t, 0, float3_as_args(n), __float_as_uint(x), __float_as_uint(y));
        }
    }
}

extern "C" __global__ void __intersection__cylinder_y()
{
    // Load shader binding table (SBT) and access data specific to this hit group
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryDataST::Cylinder_Y& cyl = sbt_data->geometry_data.getCylinder_Y();

    // Get ray information: origin, direction, and min/max distances over which ray should be tested
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = normalize(optixGetWorldRayDirection());
    const float  ray_tmin = optixGetRayTmin();
    const float  ray_tmax = optixGetRayTmax();

    // Transform ray to the cylinder's local coordinate system
    float3 local_ray_orig = ray_orig - cyl.center;
    float3 local_ray_dir = ray_dir;

	// TODO: check how to optimize this, there should be a way in optix to rotate coordinates 
    float3 local_x = cyl.base_x;
    float3 local_z = cyl.base_z;
    float3 local_y = cross(local_z, local_x);

    local_ray_orig = make_float3(
        dot(local_ray_orig, local_x),
        dot(local_ray_orig, local_y),
        dot(local_ray_orig, local_z)
    );
    local_ray_dir = make_float3(
        dot(local_ray_dir, local_x),
        dot(local_ray_dir, local_y),
        dot(local_ray_dir, local_z)
    );

	// solve quadratic equation for intersection
    float A = local_ray_dir.x * local_ray_dir.x + local_ray_dir.z * local_ray_dir.z;
    float B = 2.0f * (local_ray_orig.x * local_ray_dir.x + local_ray_orig.z * local_ray_dir.z);
    float C = local_ray_orig.x * local_ray_orig.x + local_ray_orig.z * local_ray_orig.z - cyl.radius * cyl.radius;

    float determinant = B * B - 4.0f * A * C;

    if (determinant < 0.0f)
    {
        // No intersection
        return;
    }

    // Compute intersection distances
    float t1 = (-B - sqrtf(determinant)) / (2.0f * A);
    float t2 = (-B + sqrtf(determinant)) / (2.0f * A);

    float t = t1 > 0.0f ? t1 : t2; // Use the closer valid intersection
    if (t < ray_tmin || t > ray_tmax)
    {
        // Intersection is out of bounds
        return;
    }

    // Compute intersection point in local space
    float3 local_hit_point = local_ray_orig + t * local_ray_dir;

    // Check if the hit point is within the cylinder's height bounds
    if (fabsf(local_hit_point.y) > cyl.half_height)
    {
        // If t1 is invalid, try t2
        t = t2;
        local_hit_point = local_ray_orig + t * local_ray_dir;
        if (t < ray_tmin || t > ray_tmax || fabsf(local_hit_point.y) > cyl.half_height)
        {
            return; // Both intersections are out of bounds
        }
    }

    // Compute normal in local coordinates
    float3 local_normal = normalize(make_float3(local_hit_point.x, 0.0f, local_hit_point.z));

    // Transform normal back to world coordinates
    float3 world_normal = local_normal.x * local_x + local_normal.y * local_y + local_normal.z * local_z;

    // Compute the hit point in world space
    float3 world_hit_point = ray_orig + t * ray_dir;

    // Report intersection to OptiX
    optixReportIntersection(
        t,
        0,
        float3_as_args(world_normal),
        __float_as_uint(world_hit_point.x),
        __float_as_uint(world_hit_point.y)
    );
}

// ray cylinder intersection with top and bottom caps 
// it can also be modeled as cylinder with two disks. 
extern "C" __global__ void __intersection__cylinder_y_capped()
{
    // Load shader binding table (SBT) and access data specific to this hit group
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryDataST::Cylinder_Y& cyl = sbt_data->geometry_data.getCylinder_Y();

    // Get ray information: origin, direction, and min/max distances over which ray should be tested
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = normalize(optixGetWorldRayDirection());
    const float  ray_tmin = optixGetRayTmin();
    const float  ray_tmax = optixGetRayTmax();

    // Transform ray to the cylinder's local coordinate system
    float3 local_ray_orig = ray_orig - cyl.center;
    float3 local_ray_dir = ray_dir;

    // Transform using the cylinder's local basis
    float3 local_x = cyl.base_x;
    float3 local_z = cyl.base_z;
    float3 local_y = cross(local_z, local_x);

    local_ray_orig = make_float3(
        dot(local_ray_orig, local_x),
        dot(local_ray_orig, local_y),
        dot(local_ray_orig, local_z)
    );
    local_ray_dir = make_float3(
        dot(local_ray_dir, local_x),
        dot(local_ray_dir, local_y),
        dot(local_ray_dir, local_z)
    );

    // Solve quadratic equation for intersection with curved surface
    float A = local_ray_dir.x * local_ray_dir.x + local_ray_dir.z * local_ray_dir.z;
    float B = 2.0f * (local_ray_orig.x * local_ray_dir.x + local_ray_orig.z * local_ray_dir.z);
    float C = local_ray_orig.x * local_ray_orig.x + local_ray_orig.z * local_ray_orig.z - cyl.radius * cyl.radius;

    float determinant = B * B - 4.0f * A * C;

    float t_curved = ray_tmax + 1.0f; // Initialize to invalid
    if (determinant >= 0.0f)
    {
        // Compute intersection distances
        float t1 = (-B - sqrtf(determinant)) / (2.0f * A);
        float t2 = (-B + sqrtf(determinant)) / (2.0f * A);

        // Select the closest valid intersection within bounds
        if (t1 > ray_tmin && t1 < ray_tmax && fabsf(local_ray_orig.y + t1 * local_ray_dir.y) <= cyl.half_height)
        {
            t_curved = t1;
        }
        else if (t2 > ray_tmin && t2 < ray_tmax && fabsf(local_ray_orig.y + t2 * local_ray_dir.y) <= cyl.half_height)
        {
            t_curved = t2;
        }
    }

    // Check intersection with top and bottom caps
    float t_caps = ray_tmax + 1.0f;
    {
        // Bottom cap: y = -half_height
        if (fabsf(local_ray_dir.y) > 1e-6f) // Avoid division by zero
        {
            float t = (-cyl.half_height - local_ray_orig.y) / local_ray_dir.y;
            float2 hit_point = make_float2(local_ray_orig.x + t * local_ray_dir.x,
                local_ray_orig.z + t * local_ray_dir.z);
            if (t > ray_tmin && t < ray_tmax && dot(hit_point, hit_point) <= cyl.radius * cyl.radius)
            {
                t_caps = t;
            }
        }

        // Top cap: y = +half_height
        if (fabsf(local_ray_dir.y) > 1e-6f)
        {
            float t = (cyl.half_height - local_ray_orig.y) / local_ray_dir.y;
            float2 hit_point = make_float2(local_ray_orig.x + t * local_ray_dir.x,
                local_ray_orig.z + t * local_ray_dir.z);
            if (t > ray_tmin && t < ray_tmax && dot(hit_point, hit_point) <= cyl.radius * cyl.radius)
            {
                t_caps = fminf(t_caps, t);
            }
        }
    }

    // Use the closest valid intersection
    float t = fminf(t_curved, t_caps);
    if (t >= ray_tmax || t <= ray_tmin)
    {
        return; // No valid intersection
    }

    // Compute intersection point and normal
    float3 local_hit_point = local_ray_orig + t * local_ray_dir;
    float3 local_normal;

    if (t == t_curved)
    {
        // Hit on the curved surface
        local_normal = normalize(make_float3(local_hit_point.x, 0.0f, local_hit_point.z));
    }
    else
    {
        // Hit on one of the caps
        local_normal = make_float3(0.0f, signbit(local_hit_point.y) ? -1.0f : 1.0f, 0.0f);
    }

    // Transform normal back to world coordinates
    float3 world_normal = local_normal.x * local_x + local_normal.y * local_y + local_normal.z * local_z;

    // Compute world-space hit point
    float3 world_hit_point = ray_orig + t * ray_dir;

    // Report intersection to OptiX
    optixReportIntersection(
        t,
        0, // User-defined instance ID or custom data
        float3_as_args(world_normal),
        __float_as_uint(world_hit_point.x),
        __float_as_uint(world_hit_point.y)
    );
}


// __intersection__rectangle_parabolic
//
// For a parabolic rectangle the base (flat projection) is defined by the anchor and two edges.
// In a local coordinate system (with origin at the anchor) the flat rectangle covers:
//    x in [0, L1]  and  y in [0, L2],
// where L1 and L2 are the lengths of the original edge vectors.
// The parabolic surface is given by:
//    z = (curv_x/2)*x^2 + (curv_y/2)*y^2
// and the ray (in local coordinates) is:
//    (ox,oy,oz) + t*(dx,dy,dz)
// We solve for t such that:
//    oz + t*dz = (curv_x/2)*(ox+t*dx)^2 + (curv_y/2)*(oy+t*dy)^2
// which expands into a quadratic: A*t^2 + B*t + C = 0.
// After finding the valid t, we compute the local hit (x,y) and then check that
//   0 <= x <= L1   and   0 <= y <= L2.
// Finally, we compute the surface normal from the paraboloid derivative
//    f_x = curv_x * x    and    f_y = curv_y * y,
// so that the (unnormalized) local normal is (-f_x, -f_y, 1).
//
// The local hit point is then transformed back to world space for reporting.
extern "C" __global__ void __intersection__rectangle_parabolic()
{
    // Load shader binding table (SBT) data and retrieve the parabolic rectangle.
    const soltrace::HitGroupData* sbt_data = reinterpret_cast<soltrace::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryDataST::Rectangle_Parabolic& rect = sbt_data->geometry_data.getRectangleParabolic();

    // Get ray information.
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float  ray_tmin = optixGetRayTmin();
    const float  ray_tmax = optixGetRayTmax();

    //
    // Build the local coordinate system.
    //
    // We assume that the rectangle was defined with an anchor at its corner and
    // two edge vectors. The stored rect.v1 and rect.v2 are the reciprocals:
    //     stored_v1 = original_v1 / dot(original_v1, original_v1)
    // Thus, the original edge lengths are:
    // Note rect.v1 had the size of 1/original_v1_length
    float L1 = 1.0f / length(rect.v1);
    float L2 = 1.0f / length(rect.v2);
    // And the unit edge directions are:
    float3 e1 = rect.v1 * L1; // recovers the original direction of edge 1, unit vector
    float3 e2 = rect.v2 * L2; // recovers the original direction of edge 2, unit ve
    // The flat (undeformed) rectangle's normal is:
    float3 n = normalize(cross(e2, e1));

    //
    // Transform ray into local coordinates.
    // The local coordinates (x,y,z) are defined such that:
    //   - The origin is at rect.anchor.
    //   - The x-axis is e1.
    //   - The y-axis is e2.
    //   - The z-axis is n.
    //
    // Compute the rectangle center (shifting from the lower-right corner)
    float3 rect_center = rect.anchor + (L1 / 2.0f) * e1 + (L2 / 2.0f) * e2;

    float3 d = ray_orig - rect_center;
    float ox = dot(d, e1);
    float oy = dot(d, e2);
    float oz = dot(d, n);

    float dx = dot(ray_dir, e1);
    float dy = dot(ray_dir, e2);
    float dz = dot(ray_dir, n);

    // Retrieve curvature parameters.
    const float curv_x = rect.curv_x;
    const float curv_y = rect.curv_y;

    float A = (curv_x * 0.5f) * (dx * dx) + (curv_y * 0.5f) * (dy * dy);
    float B = curv_x * (ox * dx) + curv_y * (oy * dy) - dz;
    float C = (curv_x * 0.5f) * (ox * ox) + (curv_y * 0.5f) * (oy * oy) - oz;

    float t = 0.0f;
    const float eps = 1e-12f;
    bool valid = false;

    if (fabsf(A) < eps) {
        // Degenerate (linear) case.
        t = -C / B;
        valid = (t > 0.0f);
    }
    else {
        float discr = B * B - 4.0f * A * C;
        if (discr >= 0.0f) {
            float sqrt_discr = sqrtf(discr);
            float t1 = (-B - sqrt_discr) / (2.0f * A);
            float t2 = (-B + sqrt_discr) / (2.0f * A);
            // Choose the smallest positive t.
            if (t1 > 0.0f && t1 < t2) {
                t = t1;
                valid = true;
            }
            else if (t2 > 0.0f) {
                t = t2;
                valid = true;
            }
        }
    }

    // Discard if no valid t or if t is not within the ray's bounds.
    if (!valid || t < ray_tmin || t > ray_tmax) {
        return;
    }

    //
    // Compute the local intersection coordinates.
    //
    float x_hit = ox + t * dx;
    float y_hit = oy + t * dy;
    // (Optionally, you could compute z_hit = oz + t*dz and verify it is near f(x,y).)

    //
    // Check if the hit is within the rectangle's flat bounds.
    // The parametric coordinates are:
    //    a1 = x_hit / (L1/2)   and   a2 = y_hit / (L2/2)
    //
    float a1 = x_hit / (L1/2.);
    float a2 = y_hit / (L2/2.);
    if (a1 < -1.0f || a1 > 1.0f || a2 < -1.0f || a2 > 1.0f) {
        return;
    }

    //
    // Compute the surface normal at the hit on the paraboloid.
    // The height function is:
    //    f(x,y) = (curv_x/2)*x^2 + (curv_y/2)*y^2
    // so its partial derivatives are:
    //    f_x = curv_x * x    and    f_y = curv_y * y.
    // Then the (unnormalized) local normal is:
    //    N_local = (-f_x, -f_y, 1) = ( -curv_x*x_hit, -curv_y*y_hit, 1 ).
    //
    float3 N_local = normalize(make_float3(-curv_x * x_hit,
        -curv_y * y_hit,
        1.0f));
    // Transform the normal back to world coordinates.
    float3 world_normal = normalize(N_local.x * e1 +
        N_local.y * e2 +
        N_local.z * n);

    // Compute the hit point in world space.
    float3 world_hit = ray_orig + t * ray_dir;

	printf("Intersection at (%f, %f, %f) with normal local (%f, %f, %f)\n",
		world_hit.x, world_hit.y, world_hit.z,
        N_local.x, N_local.y, N_local.z);

    // Report the intersection.
    // Here, the two reported extra attributes are the parametric coordinates (a1, a2),
    // encoded as unsigned integers.
    optixReportIntersection(t, 0,
        float3_as_args(world_normal),
        __float_as_uint(a1),
        __float_as_uint(a2));

    
}
