#include "hip/hip_runtime.h"
#include <optix.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_vector_types.h>

#include "helpers.h"
#include "random.h"
#include "Soltrace.h"

// Launch parameters for soltrace
extern "C" {
    __constant__ soltrace::LaunchParams params;
}

// Halton sequence generator, used for quasi-random sampling
// Generates a Halton sequence value for a given index and base
__device__ float halton(int index, int base) {
    float f = 1.0f, result = 0.0f;
    while (index > 0) {
        f = f / base;
        result = result + f * (index % base);
        index = index / base;
    }
    return result;
}

// Generate a sample point within a parallelogram defined by the AABB (Axis-Aligned Bounding Box)
// Uses the Halton sequence for sampling
__device__ float2 haltonSampleInParallelogram(OptixAabb aabb, int sample_index) {
    // Generate Halton sequence values
    float u = halton(sample_index, 2); // Base 2 for x
    float v = halton(sample_index, 3); // Base 3 for y

    // Map the Halton values to the parallelogram
    float sampled_x = aabb.minX + u * (aabb.maxX - aabb.minX);
    float sampled_y = aabb.minY + v * (aabb.maxY - aabb.minY);

    return make_float2(sampled_x, sampled_y);
}

// Generate a random sample point within a parallelogram using a random number generator (RNG)
// The parallelogram is defined by an AABB (Axis-Aligned Bounding Box)
__device__ float2 samplePointInParallelogram(OptixAabb aabb, unsigned int seed) {
    hiprandState rng_state;
    hiprand_init(seed, 0, 0, &rng_state);

    // Generate random values between 0 and 1
    float u = hiprand_uniform(&rng_state);
    float v = hiprand_uniform(&rng_state);

    // Interpolate between the bounds of the parallelogram
    float x = aabb.minX + u * (aabb.maxX - aabb.minX);
    float y = aabb.minY + v * (aabb.maxY - aabb.minY);

    // Return the sampled point
    return make_float2(x, y);
}

// Generate a random point within a disk with a given radius
// Uses polar coordinates (r, theta) for sampling
__device__ float2 samplePointInDisk(float radius, unsigned int seed) {
    hiprandState rng_state;
    hiprand_init(seed, 0, 0, &rng_state);

    // Generate random radius and angle values
    float r = radius * sqrtf(hiprand_uniform(&rng_state));   
    float theta = 2.0f * M_PIf * hiprand_uniform(&rng_state);

    // Convert to Cartesian coordinates
    return make_float2(r * cosf(theta), r * sinf(theta));
}

// Sample a random ray direction within a cone defined by a maximum angle
__device__ float3 sampleRayDirection(float max_angle, unsigned int seed) {
    hiprandState rng_state;
    hiprand_init(seed, 0, 0, &rng_state);

    // Sample a random angle within the cone's angular spread
    float angle = max_angle * hiprand_normal(&rng_state);   // Random angle within max angular spread
    float phi = 2.0f * M_PIf * hiprand_normal(&rng_state);  // Random azimuthal angle

    // Convert spherical coordinaes to Cartesian for ray direction
    float x = sinf(angle) * cosf(phi);
    float y = sinf(angle) * sinf(phi);
    float z = -cosf(angle); // Z is negative to ensure the cone is pointing downward (towards the scene)

    return normalize(make_float3(x, y, z));
}

// == Ray Generation Program - Sun Source (Parallelogram Sampling)
extern "C" __global__ void __raygen__sun_source()
{
    // Lookup location in launch grid
    const uint3 launch_idx = optixGetLaunchIndex();         // Index of the current launch thread
    const uint3 launch_dims = optixGetLaunchDimensions();   // Dimensions of the launch grid
    const unsigned int ray_number = launch_idx.y * launch_dims.x + launch_idx.x;  // Unique ray ID

    // TODO: add a buffer around smallest AABB
    float2 sun_sample_pos = haltonSampleInParallelogram(params.scene_aabb, ray_number);

    // Sample emission angle here - capturing sun distribution
    // TODO: this is assuming the sun is directly above the scene (sun vector (0, 0, height)) to avoid projections for now
    const float3 ray_gen_pos = params.sun_center + make_float3(sun_sample_pos.x, sun_sample_pos.y, 0.0f);
    float3 initial_ray_dir = normalize(make_float3(sun_sample_pos.x, sun_sample_pos.y, 0.0f) - ray_gen_pos);
    // Add some angular variation to the ray direction to simulate the sun's spread
    float3 ray_dir = initial_ray_dir + sampleRayDirection(params.max_sun_angle, launch_idx.x);

    // Create the PerRayData structure to track ray state (e.g., path index and recursion depth)
    soltrace::PerRayData prd;
    prd.ray_path_index = ray_number;
    prd.depth = 0;

    /*
    params.hit_point_buffer[params.max_depth * prd.ray_path_index] = make_float4(0.0f, ray_gen_pos);
    */

    // Cast and trace the ray through the scene
    optixTrace(
        params.handle,               // Acceleration structure handle
        ray_gen_pos,                 // Ray origin
        ray_dir,                     // Ray direction
        0.001f,                      // Minimum ray distance (near hit distance)
        1e16f,                       // Maximum ray distance (far hit distance)
        0.0f,                        // Time parameter (static for now)
        OptixVisibilityMask(1),      // Visibility mask (e.g., to restrict ray interactions)
        OPTIX_RAY_FLAG_NONE,         // Ray flags (no special flags)
        soltrace::RAY_TYPE_RADIANCE, // Ray type (radiance for sunlight)
        soltrace::RAY_TYPE_COUNT,    // Number of ray types
        soltrace::RAY_TYPE_RADIANCE, // SBT offset (ray type to launch)
        reinterpret_cast<unsigned int&>(prd.ray_path_index),
        reinterpret_cast<unsigned int&>(prd.depth)  
    );
}

/*
// == Ray Generation Program - Sun disk
extern "C" __global__ void __raygen__sun_source()
{
    // Lookup location in launch grid here
    const uint3 launch_idx = optixGetLaunchIndex();
    const uint3 launch_dims = optixGetLaunchDimensions();
    const unsigned int ray_number = launch_idx.y * launch_dims.x + launch_idx.x;
    const unsigned int seed = launch_idx.x; // Use launch index to seed RNG for unique sampling

    float2 sun_sample_pos = samplePointInDisk(params.sun_radius, seed);

    // Sample emission angle here - capturing sun distribution
    // TODO need to update for sun's position angle
    const float3 ray_gen_pos = params.sun_center + make_float3(sun_sample_pos.x, sun_sample_pos.y, 0.0f);

    //float3 initial_ray_dir = normalize(params.scene_position - ray_gen_pos);
    float3 initial_ray_dir = normalize(make_float3(sun_sample_pos.x, sun_sample_pos.y, 0.0f) - ray_gen_pos);
    float3 ray_dir = initial_ray_dir + sampleRayDirection(params.max_sun_angle, seed);

    soltrace::PerRayData prd;
    prd.ray_path_index = ray_number;
    prd.depth = 0;

    params.hit_point_buffer[params.max_depth * prd.ray_path_index] = make_float4(1.0f, ray_gen_pos);

    // Cast and trace the ray through the scene
    optixTrace(
        params.handle,
        ray_gen_pos,
        ray_dir,
        0.001f,
        1e16f,
        0.0f,
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        soltrace::RAY_TYPE_RADIANCE,
        soltrace::RAY_TYPE_COUNT,
        soltrace::RAY_TYPE_RADIANCE,
        reinterpret_cast<unsigned int&>(prd.ray_path_index),
        reinterpret_cast<unsigned int&>(prd.depth)
    );
}
*/


