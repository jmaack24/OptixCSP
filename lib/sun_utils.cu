#include "hip/hip_runtime.h"
#include <optix.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_vector_types.h>

#include <cuda/helpers.h>
#include <cuda/random.h>
#include <cuda/Soltrace.h>
#include <hip/hip_runtime.h>
#include <iostream>

// TODO: need to figure out native support to this, there has to be a way 
// to call atomicMax with float 
__device__ float atomicMaxFloat(float* address, float val) {
    int* address_as_int = (int*)address;
    int old = *address_as_int, assumed;

    do {
        assumed = old;
        float f = __int_as_float(assumed);
        if (f >= val) break;
        old = atomicCAS(address_as_int, assumed, __float_as_int(val));
    } while (assumed != old);

    return __int_as_float(old);
}

__device__ float atomicMinFloat(float* address, float val) {
    int* address_as_int = (int*)address;
    int old = *address_as_int, assumed;

    do {
        assumed = old;
        float f = __int_as_float(assumed);
        if (f <= val) break;
        old = atomicCAS(address_as_int, assumed, __float_as_int(val));
    } while (assumed != old);

    return __int_as_float(old);
}

__device__ inline void getAABBCornersDevice(const OptixAabb& aabb, float3 corners[8]) {
    corners[0] = make_float3(aabb.minX, aabb.minY, aabb.minZ);
    corners[1] = make_float3(aabb.maxX, aabb.minY, aabb.minZ);
	corners[2] = make_float3(aabb.minX, aabb.maxY, aabb.minZ);
	corners[3] = make_float3(aabb.maxX, aabb.maxY, aabb.minZ);
	corners[4] = make_float3(aabb.minX, aabb.minY, aabb.maxZ);
	corners[5] = make_float3(aabb.maxX, aabb.minY, aabb.maxZ);
	corners[6] = make_float3(aabb.minX, aabb.maxY, aabb.maxZ);
	corners[7] = make_float3(aabb.maxX, aabb.maxY, aabb.maxZ);
}


__global__ void calculateMaxD_Kernel(
    const OptixAabb* all_aabbs_D,
    int num_aabbs,
    float3 sun_dir_normalized,
    float* out_max_d_D)
{

    extern __shared__ float sdata[]; // shared for max D reduction

    
    unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= num_aabbs) {
        sdata[threadIdx.x] = 0.0f;
    }

    // Each thread computes the max D for its assigned AABB
	OptixAabb aabb = all_aabbs_D[thread_id];

    float3 corners[8];
    getAABBCornersDevice(aabb, corners);
	float max_d = 0.0f;

	for (int i = 0; i < 8; i++) {
		max_d = fmaxf(max_d, abs(dot(corners[i], sun_dir_normalized)));
	}

    sdata[threadIdx.x] = max_d;


    __syncthreads();


	// reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sdata[threadIdx.x] = fmaxf(sdata[threadIdx.x], sdata[threadIdx.x + s]);
        }
        __syncthreads(); //
    }

    if (threadIdx.x == 0) {
        atomicMaxFloat(out_max_d_D, sdata[0]);
    }


}

__global__ void calculateUVBounds_Kernel( const OptixAabb* all_aabbs_D,
    int num_aabbs, 
    float d_plane_distance, 
    float3 sun_vec_norm,
    float3 sun_u,
    float3 sun_v,
    float tan_sun_angle,
    float* out_uv_bounds_D // Points to [u_min, u_max, v_min, v_max] on GPU
) {

    // map aabb index to vertices
    unsigned int total_vertices = num_aabbs * 8;
    unsigned int vertex_global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (vertex_global_idx >= total_vertices) {
        return;
    }

    // Find which AABB and which corner this vertex_global_idx corresponds to
    unsigned int aabb_idx = vertex_global_idx / 8;
    unsigned int corner_idx = vertex_global_idx % 8;

    OptixAabb current_aabb = all_aabbs_D[aabb_idx];
    float3 corners[8];
    getAABBCornersDevice(current_aabb, corners);
    float3 vertex_point = corners[corner_idx];

    float dist_along_sun_axis = abs(dot(vertex_point, sun_vec_norm));
    float buffer = dist_along_sun_axis * tan_sun_angle;

    float3 plane_center = d_plane_distance * sun_vec_norm;
    float3 projected_point = vertex_point - dot(vertex_point - plane_center, sun_vec_norm) * sun_vec_norm;

    float u = dot(projected_point, sun_u);
    float v = dot(projected_point, sun_v);


    // TODO: shared memory for this? 
    atomicMinFloat(&out_uv_bounds_D[0], u - buffer);
    atomicMaxFloat(&out_uv_bounds_D[1], u + buffer);
    atomicMinFloat(&out_uv_bounds_D[2], v - buffer);
    atomicMaxFloat(&out_uv_bounds_D[3], v + buffer);
}


// == Host Wrapper for launching calculateMaxD_Kernel ==
void compute_d_on_gpu(const OptixAabb* all_aabbs_D,
                      int num_aabbs,
                      float3 sun_dir_normalized,
                      float* d_out_max_d_on_gpu) {

    float initial_d_val = 0.0f;
    hipMemcpy(d_out_max_d_on_gpu, &initial_d_val, sizeof(float), hipMemcpyHostToDevice);


    int threads_per_block = 256;
    int blocks_per_grid = (num_aabbs + threads_per_block - 1) / threads_per_block;
    size_t shared_mem_size = threads_per_block * sizeof(float); // For the reduction in the kernel

    calculateMaxD_Kernel<<<blocks_per_grid, threads_per_block, shared_mem_size>>>(all_aabbs_D, num_aabbs, sun_dir_normalized, d_out_max_d_on_gpu);
}

// wrapper for launching uv bounds kernel
void compute_uv_bounds_on_gpu(
    const OptixAabb* d_all_aabbs,
    int num_aabbs,
    float d_plane_val,
    const float3& sun_vector_normalized,
    const float3& sun_u_basis,
    const float3& sun_v_basis,
    float tan_max_angle,
    float* d_out_uv_bounds
){

    float initial_bounds[4] = { FLT_MAX, -FLT_MAX, FLT_MAX, -FLT_MAX };
    hipMemcpy(d_out_uv_bounds, initial_bounds, 4 * sizeof(float), hipMemcpyHostToDevice);

    unsigned int total_vertices = num_aabbs * 8;
    if (total_vertices == 0) return;

    int threads_per_block = 256;
    int blocks_per_grid = (total_vertices + threads_per_block - 1) / threads_per_block;

    calculateUVBounds_Kernel<<<blocks_per_grid, threads_per_block>>>(d_all_aabbs, num_aabbs, d_plane_val, sun_vector_normalized, sun_u_basis, sun_v_basis, tan_max_angle, d_out_uv_bounds);
}